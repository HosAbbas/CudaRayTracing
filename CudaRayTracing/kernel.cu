#include "hip/hip_runtime.h"


//OpenGL
#define GLEW_STATIC
#include <GL\glew.h>
#include <GLFW\glfw3.h>
#include <SOIL.h>
#include "ShaderProgram.h"

//Cuda
#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"


#include <stdio.h>
#include <iostream>

using namespace std;













//Quad utilis� pour le rendu de la texture
GLfloat quad[]
{
	-0.5f, -0.5f, 0.0f, 0.0f, 0.0f,
	-0.5f, 0.5f, 0.0f, 0.0f, 1.0f,
	0.5f, 0.5f, 0.0f, 1.0f, 1.0f,

	0.5f, 0.5f, 0.0f, 1.0f, 1.0f,
	0.5f, -0.5f, 0.0f, 1.0f, 0.0f,
	-0.5f, -0.5f, 0.0f, 0.0f, 0.0f
};


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode);


__global__ void fillColor(unsigned char* d_out)
{
	/*int t_idx = threadIdx.x;
	int bx_idx = blockIdx.x;
	int by_idx = blockIdx.y;

	d_out[by_idx * 3 * 1280 + bx_idx * 256 * 3 + t_idx * 3] = 0.5;
	d_out[by_idx * 3 * 1280 + bx_idx * 256 * 3 + t_idx * 3+1] = 0.5;
	d_out[by_idx * 3 * 1280 + bx_idx * 256 * 3 + t_idx * 3+2] = 0.5;*/

	int idx = blockIdx.x * 512*3 + threadIdx.x;
	d_out[idx] = 100;
	d_out[idx + 1] = 100;
	d_out[idx + 2] = 100;
}


int main()
{
	int ResolutionX = 1280;
	int ResolutionY = 720;

	//Initialisation d'OpenGL
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);

	GLFWwindow* window = glfwCreateWindow(ResolutionX, ResolutionY, "Cuda Lancer de rayon", nullptr, nullptr);
	if (window == nullptr)
	{
		std::cout << "FAILED TO CREATE GLFW WINDOW" << std::endl;
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);


	glewExperimental = GL_TRUE;
	cout << glewInit() << endl;
	if (glewInit() != GLEW_OK)
	{
		std::cout << "FAILED TO INITIALIZE GLEW" << std::endl;

		return -1;
	}

	int width, height;
	glfwGetFramebufferSize(window, &width, &height);
	glViewport(0, 0, width, height);
	//glEnable(GL_DEPTH_TEST);
	//glEnable(GL_BLEND);
	//glEnable(GL_STENCIL_TEST);
	glClearColor(1.0, 0.0, 0.0, 1.0);
	glfwSetKeyCallback(window, key_callback);
	//glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);




	//Cr�ation du quad pour le rendu
	//Cr�ation du VAO
	GLuint VAO;
	glGenVertexArrays(1, &VAO);
	glBindVertexArray(VAO);
	GLuint quadBufferID;
	glGenBuffers(1,&quadBufferID); //G�n�ration du Buffer
	glBindBuffer(GL_ARRAY_BUFFER, quadBufferID); //Ce buffer est attach� au Vertex Buffer
	glBufferData(GL_ARRAY_BUFFER, sizeof(quad), quad, GL_STATIC_DRAW);
	//Pour les shaders
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(GLfloat), (GLvoid*)0);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(GLfloat), (GLvoid*)(3 * sizeof(GLfloat)));
	glEnableVertexAttribArray(1);
	glBindVertexArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	//Shaders
	ShaderProgram quadShader("quadVS.vs", "quadFS.fs"); //Shaders associ�s

	//Cr�ation du context Cuda/OpenGL
	hipError_t cudaStatus;
	cudaStatus = cudaGLSetGLDevice(0); // l'argument est le num�ro de la carte graphique. Ca peut �tre 0, 1 ...
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, " cudaGLSetGLDevice failed!  Do you have a CUDA-capable GPU installed?");
		system("PAUSE");
	}
	

	//Allocation du buffer PBO et lien avec Cuda
	GLuint pixelBufferID;
	hipGraphicsResource_t cudaResourceBuff;
	glGenBuffers(1, &pixelBufferID);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pixelBufferID);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, ResolutionX*ResolutionY * 3 *sizeof(GLubyte), NULL, GL_DYNAMIC_COPY);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	cudaStatus = hipGraphicsGLRegisterBuffer(&cudaResourceBuff, pixelBufferID, hipGraphicsRegisterFlagsWriteDiscard);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, " hipGraphicsGLRegisterBuffer failed!");
		cout << hipGetErrorString(cudaStatus) << endl;
		system("PAUSE");
	}

	//Cr�ation de la texture
	GLuint textureID;
	glGenTextures(1, &textureID);
	glBindTexture(GL_TEXTURE_2D, textureID);

	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, ResolutionX, ResolutionY, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
	glGenerateMipmap(GL_TEXTURE_2D);


	//Doit �tre fait � chaque it�ration
	while (!glfwWindowShouldClose(window))
	{
		//cout << "Nouvelle boucle" << endl;
		glClearColor(1.0f, 0.1f, 0.1f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);
		

		//Blocage de la ressource pour CUDA
		//Toute tentative d'acc�s � la ressource par un autre moyen engendra une erreur
		cudaStatus = hipGraphicsMapResources(1, &cudaResourceBuff, 0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, " hipGraphicsMapResources failed!");
			system("PAUSE");
		}

		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
		unsigned char* deviceBufferPtr = 0;
		size_t size =0;
		cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&deviceBufferPtr, &size, cudaResourceBuff);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, " hipGraphicsResourceGetMappedPointer failed!");
			cout << hipGetErrorString(cudaStatus) << endl;
			cout << size << endl;
			system("PAUSE");
		}
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
		//Lancement du Kernel
		cout << "Lancement du Kernel" << endl;
		fillColor<<< 1800, 512 >>>(deviceBufferPtr);

		//Deblocage de la ressource
		cout << "Unmap" << endl;
		cudaStatus = hipGraphicsUnmapResources(1, &cudaResourceBuff, 0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, " hipGraphicsUnmapResources failed!");
			cout << hipGetErrorString(cudaStatus) << endl;
			system("PAUSE");
		}

		unsigned char h_in[100];
		cout << sizeof(char) << endl;
		hipMemcpy(h_in, deviceBufferPtr, 100, hipMemcpyDeviceToHost);
		for (int i = 0; i < 100; i++)
		{
			cout << h_in[i] << endl;
		}
		system("PAUSE");

		//Transfert du pbo � la texture
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pixelBufferID);
		glBindTexture(GL_TEXTURE_2D, textureID);
		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, ResolutionX, ResolutionY, GL_RGB, GL_UNSIGNED_BYTE, NULL);

		//Affichage
		quadShader.use();
		glBindVertexArray(VAO);
		glDrawArrays(GL_TRIANGLES, 0, 6);
		glBindVertexArray(0);
		glfwSwapBuffers(window);
		glfwPollEvents();
	}
	glfwTerminate();
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}


//void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode)
//{
//}
//
//void mouse_callback(GLFWwindow* window, double xpos, double ypos)
//{
//
//}
//
//void scroll_callback(GLFWwindow* window, double xoffset, double yoffset)
//{
//
//}

void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode)
{
	if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
	{
		glfwSetWindowShouldClose(window, GL_TRUE);
	}
}